#include "cuda_include.cuh"
hipError_t cudaStatus;
GPU_info::GPU_info() {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    data = prop;
    BLOCK_SIZE = prop.maxThreadsPerBlock;
}
void GPU_info::prt() {
    std::cout << "name" << data.name << std::endl;
    std::cout << "ability" << data.major << "." << data.minor << std::endl;
    std::cout << "maxThreadsPerBlock" << data.maxThreadsPerBlock << std::endl;
    std::cout << "maxGridSize"
        << data.maxGridSize[0] << " x "
        << data.maxGridSize[1] << " x "
        << data.maxGridSize[2] << std::endl;
    std::cout << "maxThreadsDim"
        << data.maxThreadsDim[0] << " x "
        << data.maxThreadsDim[1] << " x "
        << data.maxThreadsDim[2] << std::endl;
    std::cout << "multiProcessorCount" << data.multiProcessorCount << std::endl;
    std::cout << "maxBlocksPerMultiProcessor" << data.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "maxThreadsPerMultiProcessor" << data.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "sharedMemPerBlock" << data.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "totalGlobalMem" << data.totalGlobalMem / (1024.0 * 1024.0) << " MB" << std::endl;
}

