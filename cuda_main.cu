#include "hip/hip_runtime.h"
#include "cuda_include.cuh"
#define array_length 10
//max blocks is 1104
__global__ void oddEvenSort(double* arr, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int phase = 0; phase < n; phase++) {
        if (phase % 2 == 0) { // ���ƽ����A�B�z���Ư���
            if (tid % 2 == 0 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    double temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        else { // �_�ƽ����A�B�z�_�Ư���
            if (tid % 2 == 1 && tid + 1 < n) {
                if (arr[tid] > arr[tid + 1]) {
                    double temp = arr[tid];
                    arr[tid] = arr[tid + 1];
                    arr[tid + 1] = temp;
                }
            }
        }
        __syncthreads(); // �O�ҨC���洫��������A�i�J�U�@��
    }
}
double* cuda_main(double* mat_input) {
    GPU_info GPU;
    cout << "Run ON: " << GPU.data.name << endl;
    GPU.prt();
    double* d_arr, * matA;
    matA = (double*)malloc(array_length * sizeof(double));
    if (!matA) {
        std::cerr << "mamory malloc failed\n";
    }
    for (int i = 0; i < array_length; i++) {
        matA[i] = mat_input[i];
    }

    hipMalloc(&d_arr, array_length * sizeof(double));
    hipMemcpy(d_arr, matA, array_length * sizeof(double), hipMemcpyHostToDevice);
    int num_blocks = (array_length + GPU.BLOCK_SIZE - 1) / GPU.BLOCK_SIZE;
    oddEvenSort << <num_blocks, GPU.BLOCK_SIZE >> > (d_arr, array_length);
    hipMemcpy(matA, d_arr, array_length * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_arr);
    return matA;
}

