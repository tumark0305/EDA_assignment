#include "hip/hip_runtime.h"
﻿// CUDA kernel and function for evaluating best selector (spring-style architecture)
#include "cuda_include.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <array>
#include <vector>
#include <climits>

using namespace std;

__device__ int count_crosses(
    const int* path_a, int len_a,
    const int* path_b, int len_b
) {
    int cross = 0;
    for (int i = 0; i < len_a / 4; ++i) {
        for (int j = 0; j < len_b / 4; ++j) {
            bool match_start = path_a[i * 4 + 0] == path_b[j * 4 + 0] && path_a[i * 4 + 1] == path_b[j * 4 + 1];
            bool match_end = path_a[i * 4 + 2] == path_b[j * 4 + 2] && path_a[i * 4 + 3] == path_b[j * 4 + 3];
            if (match_end) ++cross;
        }
    }
    return cross;
}

__global__ void evaluate_selectors_kernel(
    int N,
    const int* option_flat,
    const int* option_lengths,
    const int* option_offsets,
    int max_path_len,
    int* cross_counts
) {
    int selector = blockIdx.x * blockDim.x + threadIdx.x;
    if (selector >= (1 << N)) return;

    __shared__ int local_paths[64 * 128]; // 64 nets x 128 max length

    int* thread_paths = &local_paths[threadIdx.x * N * max_path_len];

    for (int i = 0; i < N; ++i) {
        int option_id = ((selector >> i) & 1);
        int index = i * 2 + option_id;
        int offset = option_offsets[index];
        int len = option_lengths[index];
        for (int j = 0; j < len; ++j) {
            thread_paths[i * max_path_len + j] = option_flat[offset + j];
        }
    }

    int total_cross = 0;
    for (int i = 0; i < N; ++i) {
        int len_a = option_lengths[i * 2 + ((selector >> i) & 1)];
        int* path_a = &thread_paths[i * max_path_len];
        for (int j = i + 1; j < N; ++j) {
            int len_b = option_lengths[j * 2 + ((selector >> j) & 1)];
            int* path_b = &thread_paths[j * max_path_len];
            total_cross += count_crosses(path_a, len_a, path_b, len_b);
        }
    }

    cross_counts[selector] = total_cross;
}

std::vector<int> evaluate_crossing_cuda(
    std::vector<int>& option_flat,
    std::vector<int>& option_lengths,
    std::vector<int>& option_offsets,
    int N,
    int max_path_len
) {
    int selector_count = 1 << N;
    std::vector<int> cross_counts(selector_count);

    int* d_option_flat;
    int* d_option_lengths;
    int* d_option_offsets;
    int* d_cross_counts;

    hipMalloc(&d_option_flat, option_flat.size() * sizeof(int));
    hipMalloc(&d_option_lengths, option_lengths.size() * sizeof(int));
    hipMalloc(&d_option_offsets, option_offsets.size() * sizeof(int));
    hipMalloc(&d_cross_counts, selector_count * sizeof(int));

    hipMemcpy(d_option_flat, option_flat.data(), option_flat.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_option_lengths, option_lengths.data(), option_lengths.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_option_offsets, option_offsets.data(), option_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (selector_count + threads - 1) / threads;

    evaluate_selectors_kernel << <blocks, threads >> > (
        N, d_option_flat, d_option_lengths, d_option_offsets, max_path_len, d_cross_counts);

    hipMemcpy(cross_counts.data(), d_cross_counts, selector_count * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_option_flat);
    hipFree(d_option_lengths);
    hipFree(d_option_offsets);
    hipFree(d_cross_counts);

    return cross_counts;
}