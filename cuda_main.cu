#include "hip/hip_runtime.h"
#include "cuda_include.cuh"
//max blocks is 1104
__device__ int2 overlap(const simple_BlockInfo& blocka, const simple_BlockInfo& blockb) {
    int x1_a = blocka.coordinate.x;
    int x2_a = x1_a + blocka.size.x;
    int y1_a = blocka.coordinate.y;
    int y2_a = y1_a + blocka.size.y;

    int x1_b = blockb.coordinate.x;
    int x2_b = x1_b + blockb.size.x;
    int y1_b = blockb.coordinate.y;
    int y2_b = y1_b + blockb.size.y;

    int x_overlap = max(0, min(x2_a, x2_b) - max(x1_a, x1_b));
    int y_overlap = max(0, min(y2_a, y2_b) - max(y1_a, y1_b));

    return make_int2(x_overlap, y_overlap);
}

__device__ int2 compute_overlap(const simple_BlockInfo& a, const simple_BlockInfo& b) {
    int dx = max(0, min(a.coordinate.x + a.size.x, b.coordinate.x + b.size.x) - max(a.coordinate.x, b.coordinate.x));
    int dy = max(0, min(a.coordinate.y + a.size.y, b.coordinate.y + b.size.y) - max(a.coordinate.y, b.coordinate.y));
    return make_int2(dx, dy);
}

__global__ void spring_method_kernel_shared(
    simple_BlockInfo* d_block_data,
    int2* d_sum_force,
    size_t block_count,
    float* d_rand_vals,
    size_t rand_count
) {
    //int a = blockIdx.y;
    //int b = blockIdx.x * blockDim.x + threadIdx.x;
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.y * blockDim.y + threadIdx.y;

    if (a >= block_count || b >= block_count || a >= b) return;

    simple_BlockInfo blockA = d_block_data[a];
    simple_BlockInfo blockB = d_block_data[b];
    int2 overlap_size = compute_overlap(blockA, blockB);
    if (overlap_size.x > 0 && overlap_size.y > 0) {
        int2 spring_force = make_int2((overlap_size.x +1) / 2, (overlap_size.y+1 ) / 2);
        int2 min_force = spring_force;

        int thread_id = a * block_count + b;
        float rand1 = d_rand_vals[thread_id % rand_count];
        float rand2 = d_rand_vals[(thread_id + 1) % rand_count];
        //cout << rand1 << endl;
        // ����O����V�G�D�n��V�]���j���|�^
        if (rand1 < 0.8f) {
            if (spring_force.x > spring_force.y) min_force.x = 0;
            else min_force.y = 0;
        }
        else {
            if (spring_force.x < spring_force.y) min_force.x = 0;
            else min_force.y = 0;
        }

        // ����O�[����� block
        if (rand2 < 0.7f) {
            atomicAdd(&d_sum_force[a].x, min_force.x);
            atomicAdd(&d_sum_force[a].y, min_force.y);
        }
        else {
            atomicAdd(&d_sum_force[b].x, -min_force.x);
            atomicAdd(&d_sum_force[b].y, -min_force.y);
        }
        
    }
    
}


__global__ void overlap_check_kernel(simple_BlockInfo* blocks, bool* result, int count) {
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    if (a >= count) return;

    for (int b = a + 1; b < count; ++b) {
        int x1_a = blocks[a].coordinate.x;
        int x2_a = x1_a + blocks[a].size.x;
        int y1_a = blocks[a].coordinate.y;
        int y2_a = y1_a + blocks[a].size.y;

        int x1_b = blocks[b].coordinate.x;
        int x2_b = x1_b + blocks[b].size.x;
        int y1_b = blocks[b].coordinate.y;
        int y2_b = y1_b + blocks[b].size.y;

        int x_overlap = max(0, min(x2_a, x2_b) - max(x1_a, x1_b));
        int y_overlap = max(0, min(y2_a, y2_b) - max(y1_a, y1_b));

        if (x_overlap > 0 && y_overlap > 0) {
            *result = false;
        }
    }
}

vector<array<int, 2>> spring_cuda(vector< BlockInfo>& block_list_input) {
    std::vector<simple_BlockInfo> simple_block_list; 
    for (const auto& block : block_list_input) {
        simple_BlockInfo cell;
        cell.coordinate = make_int2(block.coordinate[0], block.coordinate[1]);
        cell.size = make_int2(block.size[0], block.size[1]);
        simple_block_list.push_back(cell); 
    } 
    
    size_t block_count = simple_block_list.size();
    std::vector<int2> sum_force(block_count, { 0, 0 }); 

    std::vector<float> rand_vals(1024); // ��ĳ�� hiprand ��
    for (auto& v : rand_vals) v = static_cast<float>(rand()) / RAND_MAX; 
    //for (auto& v : rand_vals) cout << v << endl;
    //cout << block_count << endl;

    float* d_rand_vals; 
    hipMalloc(&d_rand_vals, rand_vals.size() * sizeof(float)); 
    hipMemcpy(d_rand_vals, rand_vals.data(), rand_vals.size() * sizeof(float), hipMemcpyHostToDevice); 

    size_t rand_count = rand_vals.size(); // �ǵ� kernel

    // Allocate memory on device
    simple_BlockInfo* d_block_data_copy;
    int2* d_sum_force; 
    hipMalloc((void**)&d_block_data_copy, block_count * sizeof(simple_BlockInfo));
    hipMalloc((void**)&d_sum_force, block_count * sizeof(int2)); 

    // Copy data to device
    hipMemcpy(d_block_data_copy, simple_block_list.data(), block_count * sizeof(simple_BlockInfo), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_force, sum_force.data(), block_count * sizeof(int2), hipMemcpyHostToDevice); 

    // Set up block and grid sizes
    dim3 blockDim(32,32);
    dim3 gridDim((block_count + blockDim.x - 1) / blockDim.x, (block_count + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    spring_method_kernel_shared << <gridDim, blockDim >> > (
        d_block_data_copy,
        d_sum_force,
        block_count,
        d_rand_vals,
        rand_count
        );
    hipDeviceSynchronize(); 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    // Copy the result back to host
    std::vector<int2> sum_force_device(block_count); 
    hipMemcpy(sum_force_device.data(), d_sum_force, block_count * sizeof(int2), hipMemcpyDeviceToHost); 

    // Free device memory
    hipFree(d_block_data_copy);
    hipFree(d_sum_force);
    hipFree(d_rand_vals); 
    // Assign to output
    vector<array<int, 2>> output; 
    for (size_t i = 0; i < block_count; ++i) {
        output.push_back({ sum_force_device[i].x,sum_force_device[i] .y});
        //cout << "force=" << sum_force_device[i].x << "," << sum_force_device[i].y << endl;
        //cout << "force=" << simple_block_list[i].size.x << "," << simple_block_list[i].size.y << endl;
    }
    return output;
}





bool no_overlap_cuda(vector<BlockInfo>& block_list_input) {
    
    //GPU.prt();
    int count = block_list_input.size();
    if (count <= 1) return true;

    std::vector<simple_BlockInfo> simple_blocks;
    for (const auto& block : block_list_input) {
        simple_BlockInfo sb;
        sb.coordinate = make_int2(block.coordinate[0], block.coordinate[1]);
        sb.size = make_int2(block.size[0], block.size[1]);
        simple_blocks.push_back(sb);
    }

    // Allocate device memory
    simple_BlockInfo* d_blocks;
    bool* d_result;
    bool result = true;

    hipMalloc(&d_blocks, count * sizeof(simple_BlockInfo));
    hipMalloc(&d_result, sizeof(bool));
    hipMemcpy(d_blocks, simple_blocks.data(), count * sizeof(simple_BlockInfo), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(bool), hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (count + threads - 1) / threads;
    overlap_check_kernel << <blocks, threads >> > (d_blocks, d_result, count);
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(&result, d_result, sizeof(bool), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_blocks);
    hipFree(d_result);

    return result;
}




